/**********************************************************************
 * DESCRIPTION:
 *   Serial Concurrent Wave Equation - C Version
 *   This program implements the concurrent wave equation
 *********************************************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

// #define CHECK_TIME

#define MAXPOINTS 1000000
#define MAXSTEPS 1000000
#define MINPOINTS 20
#define PI 3.14159265f

void check_param(void);
void init_line(void);
void update (void);
void printfinal (void);

int nsteps,                 	/* number of time steps */
    tpoints, 	     		/* total points along string */
    rcode;                  	/* generic return code */
float  values[MAXPOINTS+2], 	/* values at time t */
       oldval[MAXPOINTS+2], 	/* values at time (t-dt) */
       newval[MAXPOINTS+2]; 	/* values at time (t+dt) */

float *dev_values, *dev_oldval, *dev_newval;

/**********************************************************************
 *	Checks input values from parameters
 *********************************************************************/
void check_param(void)
{
   char tchar[20];

   /* check number of points, number of iterations */
   while ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS)) {
      printf("Enter number of points along vibrating string [%d-%d]: "
           ,MINPOINTS, MAXPOINTS);
      scanf("%s", tchar);
      tpoints = atoi(tchar);
      if ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS))
         printf("Invalid. Please enter value between %d and %d\n", 
                 MINPOINTS, MAXPOINTS);
   }
   while ((nsteps < 1) || (nsteps > MAXSTEPS)) {
      printf("Enter number of time steps [1-%d]: ", MAXSTEPS);
      scanf("%s", tchar);
      nsteps = atoi(tchar);
      if ((nsteps < 1) || (nsteps > MAXSTEPS))
         printf("Invalid. Please enter value between 1 and %d\n", MAXSTEPS);
   }

   printf("Using points = %d, steps = %d\n", tpoints, nsteps);

}

/**********************************************************************
 *     Initialize points on line
 *********************************************************************/
void init_line(void)
{
   int i, j;
   float x, fac, k, tmp;

   /* Calculate initial values based on sine curve */
   fac = 2.0f * PI;
   k = 0.0f; 
   tmp = tpoints - 1.0f;
   for (j = 1.0f; j <= tpoints; j++) {
      x = k/tmp;
      values[j] = sin (fac * x);
      k = k + 1.0f;
   } 

   /* Initialize old values array */
   for (i = 1; i <= tpoints; i++) 
      oldval[i] = values[i];
}

/**********************************************************************
 *      Calculate new values using wave equation
 *********************************************************************/

 void move_to_device()
 {
    // load data to device memory
   hipMalloc(&dev_values, MAXPOINTS+2);
   hipMemcpy(dev_values, values, MAXPOINTS+2, hipMemcpyHostToDevice);
   hipMalloc(&dev_oldval, MAXPOINTS+2);
   hipMemcpy(dev_oldval, oldval, MAXPOINTS+2, hipMemcpyHostToDevice);
   hipMalloc(&dev_newval, MAXPOINTS+2);
   hipMemcpy(dev_newval, newval, MAXPOINTS+2, hipMemcpyHostToDevice);
 }

 void move_to_host()
 {
    // load data from GPU
   hipMemcpy(values, dev_values, MAXPOINTS+2, hipMemcpyDeviceToHost);
   hipFree(dev_newval);
   hipFree(dev_oldval);
   hipFree(dev_values);
 }

// cuda version
__global__ void DoMath(float* dev_newval, float* dev_oldval, float* dev_values)
{
   float dtime, c, dx, tau, sqtau;
   int i = threadIdx.x;

   dtime = 0.3f;
   c = 1.0f;
   dx = 1.0f;
   tau = (c * dtime / dx);
   sqtau = tau * tau;
   dev_newval[i] = (2.0f * dev_values[i]) - dev_oldval[i] + (sqtau *  (-2.0f)*dev_values[i]);
}

__global__ void UpdateOldVal(float* dev_newval, float* dev_oldval, float* dev_values)
{
   int i = threadIdx.x;
   dev_oldval[i] = dev_values[i];
   dev_values[i] = dev_newval[i];
}


 void do_math(int i)
{
   float dtime, c, dx, tau, sqtau;

   dtime = 0.3;
   c = 1.0;
   dx = 1.0;
   tau = (c * dtime / dx);
   sqtau = tau * tau;
   newval[i] = (2.0 * values[i]) - oldval[i] + (sqtau *  (-2.0)*values[i]);
}

/**********************************************************************
 *     Update all values along line a specified number of times
 *********************************************************************/
void update()
{
   int i;

   /* Update values for each time step */
   for (i = 1; i<= nsteps; i++) {
      /* Update points along line for this time step */
      // if ((j == 1) || (j  == tpoints))
      newval[1] = 0.0f;
      newval[tpoints] = 0.0f;

      DoMath<<<2,tpoints>>>(dev_newval, dev_oldval, dev_values);

      // for (j = 1; j <= tpoints; j++) {
      //    /* global endpoints */
      //    if ((j == 1) || (j  == tpoints))
      //       newval[j] = 0.0;
      //    else
      //       do_math(j);
      // }
      

      /* Update old values with new values */
      // for (j = 1; j <= tpoints; j++) {
      //    oldval[j] = values[j];
      //    values[j] = newval[j];
      // }
      UpdateOldVal<<<1,tpoints>>>(dev_newval, dev_oldval, dev_values);
   }
}

/**********************************************************************
 *     Print final results
 *********************************************************************/
void printfinal()
{
   int i;

   for (i = 1; i <= tpoints; i++) {
      printf("%6.4f ", values[i]);
      if (i%10 == 0)
         printf("\n");
   }
}

/**********************************************************************
 *	Main program
 *********************************************************************/
int main(int argc, char *argv[])
{
	sscanf(argv[1],"%d",&tpoints);
	sscanf(argv[2],"%d",&nsteps);
   check_param();
   #ifdef CHECK_TIME
   clock_t begin = clock();
   #endif
   printf("Initializing points on the line...\n");
   init_line();

   move_to_device();

	printf("Updating all points for all time steps...\n");
	update();
   printf("Printing final results...\n");
   
   move_to_host();

	printfinal();
   printf("\nDone.\n\n");
   #ifdef CHECK_TIME
	clock_t end = clock();  
   double time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
   printf("time: %4f sec\n", time_spent);
   #endif
	return 0;
}
