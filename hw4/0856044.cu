#include "hip/hip_runtime.h"
/**********************************************************************
 * DESCRIPTION:
 *   Serial Concurrent Wave Equation - C Version
 *   This program implements the concurrent wave equation
 *********************************************************************/
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#define MAXPOINTS 1000000
#define MAXSTEPS 1000000
#define MINPOINTS 20
#define PI 3.14159265

void check_param(void);
void init_line(void);
void update (void);
void printfinal (void);

int nsteps,                 	/* number of time steps */
    tpoints, 	     		/* total points along string */
    rcode;                  	/* generic return code */
float  values[MAXPOINTS+2], 	/* values at time t */
       oldval[MAXPOINTS+2], 	/* values at time (t-dt) */
       newval[MAXPOINTS+2]; 	/* values at time (t+dt) */

float *dev_values, *dev_oldval, *dev_newval;

/**********************************************************************
 *	Checks input values from parameters
 *********************************************************************/
void check_param(void)
{
   char tchar[20];

   /* check number of points, number of iterations */
   while ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS)) {
      printf("Enter number of points along vibrating string [%d-%d]: "
           ,MINPOINTS, MAXPOINTS);
      scanf("%s", tchar);
      tpoints = atoi(tchar);
      if ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS))
         printf("Invalid. Please enter value between %d and %d\n", 
                 MINPOINTS, MAXPOINTS);
   }
   while ((nsteps < 1) || (nsteps > MAXSTEPS)) {
      printf("Enter number of time steps [1-%d]: ", MAXSTEPS);
      scanf("%s", tchar);
      nsteps = atoi(tchar);
      if ((nsteps < 1) || (nsteps > MAXSTEPS))
         printf("Invalid. Please enter value between 1 and %d\n", MAXSTEPS);
   }

   printf("Using points = %d, steps = %d\n", tpoints, nsteps);

}

/**********************************************************************
 *     Initialize points on line
 *********************************************************************/
void init_line(void)
{
   int i, j;
   float x, fac, k, tmp;

   /* Calculate initial values based on sine curve */
   fac = 2.0 * PI;
   k = 0.0; 
   tmp = tpoints - 1;
   for (j = 1; j <= tpoints; j++) {
      x = k/tmp;
      values[j] = sin (fac * x);
      k = k + 1.0;
   } 

   /* Initialize old values array */
   for (i = 1; i <= tpoints; i++) 
      oldval[i] = values[i];
}

/**********************************************************************
 *      Calculate new values using wave equation
 *********************************************************************/
// cuda version

__global__ void DoMath()
{
   float dtime, c, dx, tau, sqtau;
   int i = threadIdx.x;

   dtime = 0.3;
   c = 1.0;
   dx = 1.0;
   tau = (c * dtime / dx);
   sqtau = tau * tau;
   newval[i] = (2.0 * values[i]) - oldval[i] + (sqtau *  (-2.0)*values[i]);
}

__global__ void UpdateOldVal()
{
   int i = threadIdx.x;
   oldval[i] = values[i];
   values[i] = newval[i];
}


 void do_math(int i)
{
   float dtime, c, dx, tau, sqtau;

   dtime = 0.3;
   c = 1.0;
   dx = 1.0;
   tau = (c * dtime / dx);
   sqtau = tau * tau;
   newval[i] = (2.0 * values[i]) - oldval[i] + (sqtau *  (-2.0)*values[i]);
}

/**********************************************************************
 *     Update all values along line a specified number of times
 *********************************************************************/
void update()
{
   int i, j;

   // load data to device memory
   hipMalloc(&dev_values, MAXPOINTS+2);
   hipMemcpy(dev_values, values, MAXPOINTS+2)
   hipMalloc(&dev_oldval, MAXPOINTS+2);
   hipMemcpy(dev_oldval, oldval, MAXPOINTS+2)
   hipMalloc(&dev_newval, MAXPOINTS+2);
   hipMemcpy(dev_newval, newval, MAXPOINTS+2)


   /* Update values for each time step */
   for (i = 1; i<= nsteps; i++) {
      /* Update points along line for this time step */
      // if ((j == 1) || (j  == tpoints))
      newval[1] = 0.0;
      newval[tpoints] = 0.0;

      DoMath<<<2,tpoints>>>();

      // for (j = 1; j <= tpoints; j++) {
      //    /* global endpoints */
      //    if ((j == 1) || (j  == tpoints))
      //       newval[j] = 0.0;
      //    else
      //       do_math(j);
      // }
      

      /* Update old values with new values */
      // for (j = 1; j <= tpoints; j++) {
      //    oldval[j] = values[j];
      //    values[j] = newval[j];
      // }
      UpdateOldVal<<<1,tpoints>>>
   }
}

/**********************************************************************
 *     Print final results
 *********************************************************************/
void printfinal()
{
   int i;

   for (i = 1; i <= tpoints; i++) {
      printf("%6.4f ", values[i]);
      if (i%10 == 0)
         printf("\n");
   }
}

/**********************************************************************
 *	Main program
 *********************************************************************/
int main(int argc, char *argv[])
{
	sscanf(argv[1],"%d",&tpoints);
	sscanf(argv[2],"%d",&nsteps);
	check_param();
   clock_t begin = clock();
	printf("Initializing points on the line...\n");
	init_line();
	printf("Updating all points for all time steps...\n");
	update();
	printf("Printing final results...\n");
	printfinal();
	printf("\nDone.\n\n");
	clock_t end = clock();  
   double time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
   printf("time: %4f sec\n", time_spent);
	return 0;
}
